#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAddition(float* a, float* b, float* result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

int main() {
    int size = 10000000;
    int byteSize = size * sizeof(float);

    float* hostVectorA = new float[size];
    float* hostVectorB = new float[size];
    float* hostResult = new float[size];

    for (int i = 0; i < size; ++i) {
        hostVectorA[i] = static_cast<float>(i);
        hostVectorB[i] = static_cast<float>(i * 2);
    }

    float* deviceVectorA, * deviceVectorB, * deviceResult;
    hipMalloc((void**)&deviceVectorA, byteSize);
    hipMalloc((void**)&deviceVectorB, byteSize);
    hipMalloc((void**)&deviceResult, byteSize);

    hipMemcpy(deviceVectorA, hostVectorA, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorB, hostVectorB, byteSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    vectorAddition<<<gridSize, blockSize>>>(deviceVectorA, deviceVectorB, deviceResult, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(hostResult, deviceResult, byteSize, hipMemcpyDeviceToHost);

    // for (int i = 0; i < size; ++i) {
    //     std::cout << hostVectorA[i] << " + " << hostVectorB[i] << " = " << hostResult[i] << std::endl;
    // }
    std::cout << "Elapsed Time: " << elapsedTime << " ms" << std::endl;

    delete[] hostVectorA;
    delete[] hostVectorB;
    delete[] hostResult;
    hipFree(deviceVectorA);
    hipFree(deviceVectorB);
    hipFree(deviceResult);

    return 0;
}